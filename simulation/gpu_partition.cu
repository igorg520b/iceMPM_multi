#include "hip/hip_runtime.h"
#include "gpu_partition.h"

constexpr double d = 2; // dimensions
constexpr double coeff1 = 1.4142135623730950; // sqrt((6-d)/2.);

__device__ uint8_t gpu_error_indicator;
__constant__ icy::SimParams gprms;

icy::SimParams *GPU_Partition::prms;

GPU_Partition::GPU_Partition()
{
    host_side_indenter_force_accumulator = nullptr;
    pts_array = nullptr;
    grid_array = nullptr;
    indenter_force_accumulator = nullptr;
    _vector_data_disabled_points = nullptr;
    for(int i=0;i<4;i++) point_transfer_buffer[i] = nullptr;
}

GPU_Partition::~GPU_Partition()
{
    hipSetDevice(Device);
    hipEventDestroy(eventCycleStart);
    hipEventDestroy(eventCycleStop);
    hipEventDestroy(event_grid_halo_sent[0]);
    hipEventDestroy(event_grid_halo_sent[1]);
    hipEventDestroy(event_pts_sent[0]);
    hipEventDestroy(event_pts_sent[1]);

    hipStreamDestroy(streamCompute);

    hipHostFree(host_side_indenter_force_accumulator);
    hipFree(indenter_force_accumulator);
    hipFree(pts_array);
    for(int i=0;i<4;i++) hipFree(point_transfer_buffer[i]);
    hipFree(_vector_data_disabled_points);
    hipFree(grid_array);
    spdlog::info("Destructor invoked; partition {} on device {}", PartitionID, Device);
}

void GPU_Partition::initialize(int device, int partition)
{
    this->PartitionID = partition;
    this->Device = device;
    hipSetDevice(Device);
    hipEventCreate(&eventCycleStart);
    hipEventCreate(&eventCycleStop);
    hipEventCreate(&event_grid_halo_sent[0]);
    hipEventCreate(&event_grid_halo_sent[1]);
    hipEventCreate(&event_pts_sent[0]);
    hipEventCreate(&event_pts_sent[1]);
    hipError_t err = hipStreamCreate(&streamCompute);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition initialization failure");
    initialized = true;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, Device);
    spdlog::info("Partition {}: initialized dev {}; compute {}.{}", PartitionID, Device,deviceProp.major, deviceProp.minor);
}

void GPU_Partition::allocate(unsigned n_points_capacity, unsigned grid_x_capacity)
{
    hipSetDevice(Device);

    // host-side indenter accumulator
    hipError_t err = hipHostMalloc(&host_side_indenter_force_accumulator, prms->IndenterArraySize());
    if(err!=hipSuccess) throw std::runtime_error("GPU_Partition allocate host-side buffer");

    // indenter accumulator
    err = hipMalloc(&indenter_force_accumulator, prms->IndenterArraySize());
    if(err != hipSuccess) throw std::runtime_error("cuda_allocate_arrays");

    // points
    const size_t pts_buffer_requested = sizeof(double) * n_points_capacity;
    err = hipMallocPitch(&pts_array, &nPtsPitch, pts_buffer_requested, icy::SimParams::nPtsArrays);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate");
    nPtsPitch /= sizeof(double);

    // point transfer buffers
    VectorCapacity_transfer = n_points_capacity * prms->PointsTransferBufferFraction;
    for(int i=0;i<4;i++)
    {
        err = hipMalloc(&point_transfer_buffer[i], VectorCapacity_transfer*icy::SimParams::nPtsArrays*sizeof(double));
        if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate");
    }

    // integer vector for disabled points
    VectorCapacity_disabled = n_points_capacity * prms->ExtraSpaceForIncomingPoints;
    err = hipMalloc(&_vector_data_disabled_points, VectorCapacity_disabled*sizeof(unsigned));
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate _vector_data_disabled_points");

    // grid
    size_t grid_size_local_requested = prms->GridY*(grid_x_capacity + 4*prms->GridHaloSize) * sizeof(double);
    err = hipMallocPitch (&grid_array, &nGridPitch, grid_size_local_requested, icy::SimParams::nGridArrays);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate grid array");
    nGridPitch /= sizeof(double); // assume that this divides without remainder

    spdlog::info("Partition {}-{}: allocated GridPitch {} ({}); Pts {}; Disabled {}; PtsTransfer {}",
                 PartitionID, Device, nGridPitch, nGridPitch/prms->GridY, nPtsPitch, VectorCapacity_disabled, VectorCapacity_transfer);
}

void GPU_Partition::update_constants()
{
    hipSetDevice(Device);
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_error_indicator), &error_code, sizeof(error_code));
    if(err != hipSuccess) throw std::runtime_error("gpu_error_indicator initialization");
    err = hipMemcpyToSymbol(HIP_SYMBOL(gprms), prms, sizeof(icy::SimParams));
    if(err!=hipSuccess) throw std::runtime_error("cuda_update_constants: gprms");
    spdlog::info("Constant symbols copied to device {}; partition {}", Device, PartitionID);
}
