#include "hip/hip_runtime.h"
#include "gpu_partition.h"

constexpr double d = 2; // dimensions
constexpr double coeff1 = 1.4142135623730950; // sqrt((6-d)/2.);

__device__ uint8_t gpu_error_indicator;
__constant__ icy::SimParams gprms;

icy::SimParams *GPU_Partition::prms;

void GPU_Partition::transfer_points_from_soa_to_device(HostSideSOA &hssoa, unsigned point_idx_offset)
{
    hipError_t err;
    err = hipSetDevice(Device);
    if(err != hipSuccess)
    {
        spdlog::critical("error setting the device {} in transfer points",Device);
        throw std::runtime_error("transfer_points_from_soa_to_device");
    }

    // due to the layout of host-side SOA, we transfer the pts arrays one-by-one
    for(int i=0;i<icy::SimParams::nPtsArrays;i++)
    {
        double *ptr_dst = pts_array + i*nPtsPitch;
        double *ptr_src = hssoa.getPointerToLine(i)+point_idx_offset;
        err = hipMemcpyAsync(ptr_dst, ptr_src, nPts_partition*sizeof(double), hipMemcpyHostToDevice, streamCompute);
        if(err != hipSuccess)
        {
            const char* errorString = hipGetErrorString(err);
            spdlog::critical("PID {}; line {}; nPts_partition {}, cuda error: {}",PartitionID, i, nPts_partition, errorString);
            throw std::runtime_error("transfer_points_from_soa_to_device");
        }
    }
}


GPU_Partition::GPU_Partition()
{
    nPts_partition = GridX_partition = GridX_offset = 0;
    host_side_indenter_force_accumulator = nullptr;
    pts_array = nullptr;
    grid_array = nullptr;
    indenter_force_accumulator = nullptr;
    _vector_data_disabled_points = nullptr;
    for(int i=0;i<4;i++) point_transfer_buffer[i] = nullptr;
}

GPU_Partition::~GPU_Partition()
{
    hipSetDevice(Device);
    hipEventDestroy(eventCycleStart);
    hipEventDestroy(eventCycleStop);
    hipEventDestroy(event_grid_halo_sent[0]);
    hipEventDestroy(event_grid_halo_sent[1]);
    hipEventDestroy(event_pts_sent[0]);
    hipEventDestroy(event_pts_sent[1]);

    hipStreamDestroy(streamCompute);

    hipHostFree(host_side_indenter_force_accumulator);
    hipFree(indenter_force_accumulator);
    hipFree(pts_array);
    for(int i=0;i<4;i++) hipFree(point_transfer_buffer[i]);
    hipFree(_vector_data_disabled_points);
    hipFree(grid_array);
    spdlog::info("Destructor invoked; partition {} on device {}", PartitionID, Device);
}

void GPU_Partition::initialize(int device, int partition)
{
    this->PartitionID = partition;
    this->Device = device;
    hipSetDevice(Device);
    hipEventCreate(&eventCycleStart);
    hipEventCreate(&eventCycleStop);
    hipEventCreate(&event_grid_halo_sent[0]);
    hipEventCreate(&event_grid_halo_sent[1]);
    hipEventCreate(&event_pts_sent[0]);
    hipEventCreate(&event_pts_sent[1]);
    hipError_t err = hipStreamCreate(&streamCompute);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition initialization failure");
    initialized = true;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, Device);
    spdlog::info("Partition {}: initialized dev {}; compute {}.{}", PartitionID, Device,deviceProp.major, deviceProp.minor);
}


void GPU_Partition::allocate(unsigned n_points_capacity, unsigned grid_x_capacity)
{
    hipSetDevice(Device);

    // host-side indenter accumulator
    hipError_t err = hipHostMalloc(&host_side_indenter_force_accumulator, prms->IndenterArraySize());
    if(err!=hipSuccess) throw std::runtime_error("GPU_Partition allocate host-side buffer");
    memset(host_side_indenter_force_accumulator, 0, prms->IndenterArraySize());

    // indenter accumulator
    err = hipMalloc(&indenter_force_accumulator, prms->IndenterArraySize());
    if(err != hipSuccess) throw std::runtime_error("cuda_allocate_arrays");

    // points
    const size_t pts_buffer_requested = sizeof(double) * n_points_capacity;
    err = hipMallocPitch(&pts_array, &nPtsPitch, pts_buffer_requested, icy::SimParams::nPtsArrays);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate");
    nPtsPitch /= sizeof(double);

    // point transfer buffers
    VectorCapacity_transfer = n_points_capacity * prms->PointsTransferBufferFraction;
    for(int i=0;i<4;i++)
    {
        err = hipMalloc(&point_transfer_buffer[i], (1+VectorCapacity_transfer*icy::SimParams::nPtsArrays)*sizeof(double));
        if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate");
    }

    // integer vector for disabled points
    VectorCapacity_disabled = n_points_capacity * prms->ExtraSpaceForIncomingPoints;
    err = hipMalloc(&_vector_data_disabled_points, (VectorCapacity_disabled+1)*sizeof(unsigned));
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate _vector_data_disabled_points");

    // grid
    size_t grid_size_local_requested = prms->GridY*(grid_x_capacity + 4*prms->GridHaloSize) * sizeof(double);
    err = hipMallocPitch (&grid_array, &nGridPitch, grid_size_local_requested, icy::SimParams::nGridArrays);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate grid array");
    nGridPitch /= sizeof(double); // assume that this divides without remainder

    spdlog::info("Partition {}-{}: allocated GridPitch {} ({}); Pts {}; Disabled {}; PtsTransfer {}; grid_size_local_requested {}",
                 PartitionID, Device, nGridPitch, nGridPitch/prms->GridY, nPtsPitch, VectorCapacity_disabled, VectorCapacity_transfer, grid_size_local_requested);
}


void GPU_Partition::clear_utility_vectors()
{
    spdlog::info("P {} D {}, utility vectors clear",PartitionID,Device);
    hipSetDevice(Device);
    hipError_t err = hipMemsetAsync(_vector_data_disabled_points, 0, sizeof(unsigned), streamCompute);
    if(err != hipSuccess) throw std::runtime_error("initialize_utility_vectors");
    for(int i=0;i<4;i++)
    {
        hipError_t err = hipMemsetAsync(point_transfer_buffer[i], 0, sizeof(double), streamCompute);
        if(err != hipSuccess) throw std::runtime_error("initialize_utility_vectors");
    }
}


void GPU_Partition::update_constants()
{
    hipSetDevice(Device);
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_error_indicator), &error_code, sizeof(error_code));
    if(err != hipSuccess) throw std::runtime_error("gpu_error_indicator initialization");
    err = hipMemcpyToSymbol(HIP_SYMBOL(gprms), prms, sizeof(icy::SimParams));
    if(err!=hipSuccess) throw std::runtime_error("cuda_update_constants: gprms");
    spdlog::info("Constant symbols copied to device {}; partition {}", Device, PartitionID);
}


void GPU_Partition::reset_grid()
{
    hipSetDevice(Device);

    size_t gridArraySize = nGridPitch * icy::SimParams::nGridArrays * sizeof(double);
    hipError_t err = hipMemsetAsync(grid_array, 0, gridArraySize, streamCompute);
    if(err != hipSuccess)
    {
        const char* errorString = hipGetErrorString(err);
        spdlog::critical("P {}; cuda_reset_grid error: {}",PartitionID, errorString);
        spdlog::critical("nGridPitch {}; GridY {}; gridArraySize {}", nGridPitch, prms->GridY, gridArraySize);
        throw std::runtime_error("cuda_reset_grid error");
    }
}


void GPU_Partition::reset_indenter_force_accumulator()
{
    hipSetDevice(Device);
    hipError_t err = hipMemsetAsync(indenter_force_accumulator, 0, prms->IndenterArraySize(), streamCompute);
    if(err != hipSuccess) throw std::runtime_error("cuda_reset_grid error");
}
