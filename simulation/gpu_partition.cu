#include "hip/hip_runtime.h"
#include "gpu_partition.h"

using namespace Eigen;

constexpr double d = 2; // dimensions
constexpr double coeff1 = 1.4142135623730950; // sqrt((6-d)/2.);

__device__ uint8_t gpu_error_indicator;
__constant__ icy::SimParams gprms;

icy::SimParams *GPU_Partition::prms;


void GPU_Partition::update_nodes()
{
    hipSetDevice(Device);
    const int nGridNodes = prms->GridY * GridX_partition;
    int tpb = prms->tpb_Upd;
    int nBlocks = (nGridNodes + tpb - 1) / tpb;
    Eigen::Vector2d ind_center(prms->indenter_x, prms->indenter_y);

    partition_kernel_update_nodes<<<nBlocks, tpb, 0, streamCompute>>>(ind_center, nGridNodes, GridX_offset,
        nGridPitch, grid_array, indenter_force_accumulator);
    if(hipGetLastError() != hipSuccess) throw std::runtime_error("cuda_update_nodes");
}



__global__ void partition_kernel_update_nodes(const Eigen::Vector2d indCenter,
    const unsigned nNodes, const unsigned gridX_offset, const unsigned pitch_grid,
                                              double *_buffer_grid, double *indenter_force_accumulator)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= nNodes) return;

    double *buffer_grid = _buffer_grid + 3*gprms.GridY*gprms.GridHaloSize;    // actual grid buffer comes after 3x halo regions
    double mass = buffer_grid[idx];
    if(mass == 0) return;

    const double &gravity = gprms.Gravity;
    const double &indRsq = gprms.IndRSq;
    const double &dt = gprms.InitialTimeStep;
    const double &ind_velocity = gprms.IndVelocity;
    const double &cellsize = gprms.cellsize;
    const double &vmax = gprms.vmax;
    const double &vmax_squared = gprms.vmax_squared;
    const unsigned &gridY = gprms.GridY;
    const unsigned &gridXTotal = gprms.GridXTotal;

    const Vector2d vco(ind_velocity,0);  // velocity of the collision object (indenter)

    Vector2d velocity(buffer_grid[1*pitch_grid + idx], buffer_grid[2*pitch_grid + idx]);
    velocity /= mass;
    velocity[1] -= gprms.dt_Gravity;
    if(velocity.squaredNorm() > vmax_squared) velocity = velocity.normalized()*vmax;

    Vector2i gi(idx/gridY + gridX_offset, idx%gridY);   // integer x-y index of the grid node
    Vector2d gnpos = gi.cast<double>()*cellsize;    // position of the grid node in the whole grid

    // indenter
    Vector2d n = gnpos - indCenter;
    if(n.squaredNorm() < indRsq)
    {
        // grid node is inside the indenter
        Vector2d vrel = velocity - vco;
        n.normalize();
        double vn = vrel.dot(n);   // normal component of the velocity
        if(vn < 0)
        {
            Vector2d vt = vrel - n*vn;   // tangential portion of relative velocity
            Vector2d prev_velocity = velocity;
            velocity = vco + vt;

            // force on the indenter
            Vector2d force = (prev_velocity-velocity)*mass/dt;
            float angle = atan2f((float)n[0],(float)n[1]);
            angle += icy::SimParams::pi;
            angle *= gprms.n_indenter_subdivisions/ (2*icy::SimParams::pi);
            int index = min(max((int)angle, 0), gprms.n_indenter_subdivisions-1);
            atomicAdd(&indenter_force_accumulator[0+2*index], force[0]);
            atomicAdd(&indenter_force_accumulator[1+2*index], force[1]);
        }
    }

    // attached bottom layer
    if(gi.y() <= 2) velocity.setZero();
    else if(gi.y() >= gridY-3 && velocity[1]>0) velocity[1] = 0;
    if(gi.x() <= 2 && velocity[0]<0) velocity[0] = 0;
    else if(gi.x() >= gridXTotal-3 && velocity[0]>0) velocity[0] = 0;

    // side boundary conditions
    //    int blocksGridX = gprms.BlockLength*gprms.cellsize_inv+5-2;
    //    int blocksGridY = gprms.BlockHeight/2*gprms.cellsize_inv+2;
    //    if(idx_x >= blocksGridX && idx_x <= blocksGridX + 2 && idx_y < blocksGridY) velocity.setZero();
    //    if(idx_x <= 7 && idx_x > 4 && idx_y < blocksGridY) velocity.setZero();

    // write the updated grid velocity back to memory
    buffer_grid[1*pitch_grid + idx] = velocity[0];
    buffer_grid[2*pitch_grid + idx] = velocity[1];
}




void GPU_Partition::receive_halos()
{
    hipSetDevice(Device);
    const unsigned haloElementCount = prms->GridHaloSize*prms->GridY;
    const unsigned tpb = 512;   // threads per block
    const unsigned blocksPerGrid = (haloElementCount + tpb - 1) / tpb;
    partition_kernel_receive_halos<<<blocksPerGrid, tpb, 0, streamCompute>>>(haloElementCount, GridX_partition, nGridPitch, grid_array);
    if(hipGetLastError() != hipSuccess) throw std::runtime_error("receive_halos kernel execution");
}


__global__ void partition_kernel_receive_halos(const unsigned haloElementCount,
                                               const unsigned gridX, const unsigned pitch_grid, double *buffer_grid)
{
    const unsigned &halo = gprms.GridHaloSize;
    const unsigned &gridY = gprms.GridY;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx >= haloElementCount) return;
    for(int i=0; i<icy::SimParams::nGridArrays; i++)
    {
        // left halo
        buffer_grid[idx + i*pitch_grid + 3*halo*gridY] += buffer_grid[idx + i*pitch_grid + 0*halo*gridY];
        // right halo
        buffer_grid[idx + i*pitch_grid + (2*halo+gridX)*gridY] += buffer_grid[idx + i*pitch_grid + 1*halo*gridY];
    }
}




double* GPU_Partition::getHaloAddress(int whichHalo, int whichGridArray)
{
    if(whichHalo == 0)
    {
        // left halo
        return grid_array + (prms->GridY * prms->GridHaloSize*2) + whichGridArray*nGridPitch;
    }
    else if(whichHalo == 1)
    {
        // right halo
        return grid_array + prms->GridY * (GridX_partition + 3*prms->GridHaloSize) + whichGridArray*nGridPitch;
    }
    else throw std::runtime_error("getHaloAddress");
}

double* GPU_Partition::getHaloReceiveAddress(int whichHalo, int whichGridArray)
{
    if(whichHalo == 0)
    {
        // left halo
        return grid_array + (prms->GridY * prms->GridHaloSize*0) + whichGridArray*nGridPitch;
    }
    else if(whichHalo == 1)
    {
        // right halo
        return grid_array + (prms->GridY * prms->GridHaloSize*1) + whichGridArray*nGridPitch;
    }
    else throw std::runtime_error("getHaloReceiveAddress");
}



void GPU_Partition::transfer_points_from_soa_to_device(HostSideSOA &hssoa, unsigned point_idx_offset)
{
    hipError_t err;
    err = hipSetDevice(Device);
    if(err != hipSuccess)
    {
        spdlog::critical("error setting the device {} in transfer points",Device);
        throw std::runtime_error("transfer_points_from_soa_to_device");
    }

    // due to the layout of host-side SOA, we transfer the pts arrays one-by-one
    for(int i=0;i<icy::SimParams::nPtsArrays;i++)
    {
        double *ptr_dst = pts_array + i*nPtsPitch;
        double *ptr_src = hssoa.getPointerToLine(i)+point_idx_offset;
        err = hipMemcpyAsync(ptr_dst, ptr_src, nPts_partition*sizeof(double), hipMemcpyHostToDevice, streamCompute);
        if(err != hipSuccess)
        {
            const char* errorString = hipGetErrorString(err);
            spdlog::critical("PID {}; line {}; nPts_partition {}, cuda error: {}",PartitionID, i, nPts_partition, errorString);
            throw std::runtime_error("transfer_points_from_soa_to_device");
        }
    }
}


GPU_Partition::GPU_Partition()
{
    nPts_partition = GridX_partition = GridX_offset = 0;
    host_side_indenter_force_accumulator = nullptr;
    pts_array = nullptr;
    grid_array = nullptr;
    indenter_force_accumulator = nullptr;
    _vector_data_disabled_points = nullptr;
    for(int i=0;i<4;i++) point_transfer_buffer[i] = nullptr;
}

GPU_Partition::~GPU_Partition()
{
    hipSetDevice(Device);
    hipEventDestroy(eventCycleStart);
    hipEventDestroy(eventCycleStop);
    hipEventDestroy(event_grid_halo_sent[0]);
    hipEventDestroy(event_grid_halo_sent[1]);
    hipEventDestroy(event_pts_sent[0]);
    hipEventDestroy(event_pts_sent[1]);

    hipStreamDestroy(streamCompute);

    hipHostFree(host_side_indenter_force_accumulator);
    hipFree(indenter_force_accumulator);
    hipFree(pts_array);
    for(int i=0;i<4;i++) hipFree(point_transfer_buffer[i]);
    hipFree(_vector_data_disabled_points);
    hipFree(grid_array);
    spdlog::info("Destructor invoked; partition {} on device {}", PartitionID, Device);
}

void GPU_Partition::initialize(int device, int partition)
{
    this->PartitionID = partition;
    this->Device = device;
    hipSetDevice(Device);
    hipEventCreate(&eventCycleStart);
    hipEventCreate(&eventCycleStop);
    hipEventCreate(&event_grid_halo_sent[0]);
    hipEventCreate(&event_grid_halo_sent[1]);
    hipEventCreate(&event_pts_sent[0]);
    hipEventCreate(&event_pts_sent[1]);
    hipError_t err = hipStreamCreate(&streamCompute);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition initialization failure");
    initialized = true;

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, Device);
    spdlog::info("Partition {}: initialized dev {}; compute {}.{}", PartitionID, Device,deviceProp.major, deviceProp.minor);
}


void GPU_Partition::allocate(unsigned n_points_capacity, unsigned grid_x_capacity)
{
    hipSetDevice(Device);

    // host-side indenter accumulator
    hipError_t err = hipHostMalloc(&host_side_indenter_force_accumulator, prms->IndenterArraySize());
    if(err!=hipSuccess) throw std::runtime_error("GPU_Partition allocate host-side buffer");
    memset(host_side_indenter_force_accumulator, 0, prms->IndenterArraySize());

    // indenter accumulator
    err = hipMalloc(&indenter_force_accumulator, prms->IndenterArraySize());
    if(err != hipSuccess) throw std::runtime_error("cuda_allocate_arrays");

    // points
    const size_t pts_buffer_requested = sizeof(double) * n_points_capacity;
    err = hipMallocPitch(&pts_array, &nPtsPitch, pts_buffer_requested, icy::SimParams::nPtsArrays);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate");
    nPtsPitch /= sizeof(double);

    // point transfer buffers
    VectorCapacity_transfer = n_points_capacity * prms->PointsTransferBufferFraction;
    for(int i=0;i<4;i++)
    {
        err = hipMalloc(&point_transfer_buffer[i], (1+VectorCapacity_transfer*icy::SimParams::nPtsArrays)*sizeof(double));
        if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate");
    }

    // integer vector for disabled points
    VectorCapacity_disabled = n_points_capacity * prms->ExtraSpaceForIncomingPoints;
    err = hipMalloc(&_vector_data_disabled_points, (VectorCapacity_disabled+1)*sizeof(unsigned));
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate _vector_data_disabled_points");

    // grid
    size_t grid_size_local_requested = prms->GridY*(grid_x_capacity + 4*prms->GridHaloSize) * sizeof(double);
    err = hipMallocPitch (&grid_array, &nGridPitch, grid_size_local_requested, icy::SimParams::nGridArrays);
    if(err != hipSuccess) throw std::runtime_error("GPU_Partition allocate grid array");
    nGridPitch /= sizeof(double); // assume that this divides without remainder

    spdlog::info("Partition {}-{}: allocated GridPitch {} ({}); Pts {}; Disabled {}; PtsTransfer {}; grid_size_local_requested {}",
                 PartitionID, Device, nGridPitch, nGridPitch/prms->GridY, nPtsPitch, VectorCapacity_disabled, VectorCapacity_transfer, grid_size_local_requested);
}


void GPU_Partition::clear_utility_vectors()
{
    spdlog::info("P {} D {}, utility vectors clear",PartitionID,Device);
    hipSetDevice(Device);
    hipError_t err = hipMemsetAsync(_vector_data_disabled_points, 0, sizeof(unsigned), streamCompute);
    if(err != hipSuccess) throw std::runtime_error("initialize_utility_vectors");
    for(int i=0;i<4;i++)
    {
        hipError_t err = hipMemsetAsync(point_transfer_buffer[i], 0, sizeof(double), streamCompute);
        if(err != hipSuccess) throw std::runtime_error("initialize_utility_vectors");
    }
}


void GPU_Partition::update_constants()
{
    hipSetDevice(Device);
    hipError_t err = hipMemcpyToSymbol(HIP_SYMBOL(gpu_error_indicator), &error_code, sizeof(error_code));
    if(err != hipSuccess) throw std::runtime_error("gpu_error_indicator initialization");
    err = hipMemcpyToSymbol(HIP_SYMBOL(gprms), prms, sizeof(icy::SimParams));
    if(err!=hipSuccess) throw std::runtime_error("cuda_update_constants: gprms");
    spdlog::info("Constant symbols copied to device {}; partition {}", Device, PartitionID);
}


void GPU_Partition::reset_grid()
{
    hipSetDevice(Device);

    size_t gridArraySize = nGridPitch * icy::SimParams::nGridArrays * sizeof(double);
    hipError_t err = hipMemsetAsync(grid_array, 0, gridArraySize, streamCompute);
    if(err != hipSuccess)
    {
        const char* errorString = hipGetErrorString(err);
        spdlog::critical("P {}; cuda_reset_grid error: {}",PartitionID, errorString);
        spdlog::critical("nGridPitch {}; GridY {}; gridArraySize {}", nGridPitch, prms->GridY, gridArraySize);
        throw std::runtime_error("cuda_reset_grid error");
    }
}


void GPU_Partition::reset_indenter_force_accumulator()
{
    hipSetDevice(Device);
    hipError_t err = hipMemsetAsync(indenter_force_accumulator, 0, prms->IndenterArraySize(), streamCompute);
    if(err != hipSuccess) throw std::runtime_error("cuda_reset_grid error");
}


void GPU_Partition::p2g()
{
    hipSetDevice(Device);
    const unsigned &n = nPts_partition;
    const unsigned &tpb = prms->tpb_P2G;
    const unsigned blocksPerGrid = (n + tpb - 1) / tpb;
    partition_kernel_p2g<<<blocksPerGrid, tpb, 0, streamCompute>>>(GridX_partition, GridX_offset, nGridPitch,
                         nPts_partition, nPtsPitch, pts_array, grid_array);
    if(hipGetLastError() != hipSuccess) throw std::runtime_error("p2g kernel");
}


__global__ void partition_kernel_p2g(const unsigned gridX, const unsigned gridX_offset, const unsigned pitch_grid,
                              const unsigned count_pts, const unsigned pitch_pts,
                              const double *buffer_pts, double *buffer_grid)
{
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(pt_idx >= count_pts) return;

    const long long* ptr = reinterpret_cast<const long long*>(&buffer_pts[pitch_pts*icy::SimParams::idx_utility_data]);
    long long utility_data = ptr[pt_idx];
    if(utility_data & 0x20000) return; // point is disabled

    const double &dt = gprms.InitialTimeStep;
    const double &vol = gprms.ParticleVolume;
    const double &h = gprms.cellsize;
    const double &h_inv = gprms.cellsize_inv;
    const double &Dinv = gprms.Dp_inv;
    const double &particle_mass = gprms.ParticleMass;

    const unsigned &gridY = gprms.GridY;
    const int &halo = gprms.GridHaloSize;

    // pull point data from SOA
    Vector2d pos, velocity;
    Matrix2d Bp, Fe;

    for(int i=0; i<icy::SimParams::dim; i++)
    {
        pos[i] = buffer_pts[pt_idx + pitch_pts*(icy::SimParams::posx+i)];
        velocity[i] = buffer_pts[pt_idx + pitch_pts*(icy::SimParams::velx+i)];
        for(int j=0; j<icy::SimParams::dim; j++)
        {
            Fe(i,j) = buffer_pts[pt_idx + pitch_pts*(icy::SimParams::Fe00 + i*icy::SimParams::dim + j)];
            Bp(i,j) = buffer_pts[pt_idx + pitch_pts*(icy::SimParams::Bp00 + i*icy::SimParams::dim + j)];
        }
    }

    Matrix2d PFt = KirchhoffStress_Wolper(Fe);
    Matrix2d subterm2 = particle_mass*Bp - (gprms.dt_vol_Dpinv)*PFt;

    Eigen::Vector2i base_coord_i = (pos*h_inv - Vector2d::Constant(0.5)).cast<int>(); // coords of base grid node for point
    Vector2d base_coord = base_coord_i.cast<double>();
    Vector2d fx = pos*h_inv - base_coord;

    // optimized method of computing the quadratic (!) weight function (no conditional operators)
    Array2d arr_v0 = 1.5-fx.array();
    Array2d arr_v1 = fx.array() - 1.0;
    Array2d arr_v2 = fx.array() - 0.5;
    Array2d ww[3] = {0.5*arr_v0*arr_v0, 0.75-arr_v1*arr_v1, 0.5*arr_v2*arr_v2};

    for (int i = 0; i < 3; i++)
        for (int j = 0; j < 3; j++)
        {
            double Wip = ww[i][0]*ww[j][1];
            Vector2d dpos((i-fx[0])*h, (j-fx[1])*h);
            Vector2d incV = Wip*(velocity*particle_mass + subterm2*dpos);
            double incM = Wip*particle_mass;

            // the x-index of the cell takes into accout the partition's offset of the gird fragment
            int i2 = i+base_coord_i[0]-gridX_offset;
            int j2 = j+base_coord_i[1];
            if(i2<(-halo) || j2<0 || i2>=(gridX+halo) || j2>=gridY) gpu_error_indicator = 1;
            int idx_gridnode = j2 + (i2+halo*3)*gridY;  // two halo lines are reserved for the incoming halo data

            // Udpate mass, velocity and force
            atomicAdd(&buffer_grid[0*pitch_grid + idx_gridnode], incM);
            atomicAdd(&buffer_grid[1*pitch_grid + idx_gridnode], incV[0]);
            atomicAdd(&buffer_grid[2*pitch_grid + idx_gridnode], incV[1]);
        }
}


__device__ Matrix2d KirchhoffStress_Wolper(const Matrix2d &F)
{
    const double &kappa = gprms.kappa;
    const double &mu = gprms.mu;

    // Kirchhoff stress as per Wolper (2019)
    double Je = F.determinant();
    Matrix2d b = F*F.transpose();
    Matrix2d PFt = mu*(1/Je)*dev(b) + kappa*(Je*Je-1.)*Matrix2d::Identity();
    return PFt;
}

// deviatoric part of a diagonal matrix
__device__ Vector2d dev_d(Vector2d Adiag)
{
    return Adiag - Adiag.sum()/2*Vector2d::Constant(1.);
}

__device__ Eigen::Matrix2d dev(Eigen::Matrix2d A)
{
    return A - A.trace()/2*Eigen::Matrix2d::Identity();
}

